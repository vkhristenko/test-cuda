/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

template<typename T>
struct Point {
    T x;
//    T y[1];
};

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

template <typename T>
__global__ 
void compute(Point<T>* a)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  a[idx].x = a[idx].x + 1;
}

template<typename T>
__global__
void compute(T* a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    a[idx] = a[idx] + 1;
}

template <typename T>
void runTestSimple(int deviceId, int n)
{
  int blockSize = 256;
  float ms;

  Point<T> *d_a;
  hipEvent_t startEvent, stopEvent;
    
//  int n = nMB*1024*1024/sizeof(Point<T>);
  
  int nEffectiveMB = n * sizeof(T) / (1024*1024);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * sizeof(T)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Offset, Bandwidth (GB/s):\n");
  
  compute<<<n/blockSize, blockSize>>>(d_a); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    compute<<<n/blockSize, blockSize>>>(d_a);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nEffectiveMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}

template <typename T>
void runTest(int deviceId, int n)
{
  int blockSize = 256;
  float ms;

  Point<T> *d_a;
  hipEvent_t startEvent, stopEvent;
    
//  int n = nMB*1024*1024/sizeof(Point<T>);
  int nEffectiveMB = n * sizeof(T) / (1024*1024);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * sizeof(Point<T>)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Offset, Bandwidth (GB/s):\n");
  
  compute<<<n/blockSize, blockSize>>>(d_a); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(Point<T>)) );

    checkCuda( hipEventRecord(startEvent,0) );
    compute<<<n/blockSize, blockSize>>>(d_a);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nEffectiveMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int n = 1000000;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  checkCuda( hipSetDevice(deviceId) )
  ;
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", n);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");
  
  if (bFp64) runTest<double>(deviceId, n);
  else       runTest<float>(deviceId, n);
  if (bFp64) runTestSimple<double>(deviceId, n);
  else       runTestSimple<float>(deviceId, n);
}
